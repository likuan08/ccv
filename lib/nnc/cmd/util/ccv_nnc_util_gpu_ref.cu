extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

static int _ccv_nnc_data_transfer(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, const ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == output_size);
	int i;
	for (i = 0; i < input_size; i++)
	{
		const ccv_nnc_tensor_t* a = inputs[i];
		assert(!CCV_IS_TENSOR_VIEW(a));
		ccv_nnc_tensor_t* b = outputs[i];
		assert(!CCV_IS_TENSOR_VIEW(b));
		assert(ccv_nnc_tensor_count(a->info) == ccv_nnc_tensor_count(b->info));
		// Assume it is 32f.
		assert(CCV_GET_DATA_TYPE(a->type) == CCV_32F);
		assert(CCV_GET_DATA_TYPE(b->type) == CCV_32F);
		size_t size = ccv_nnc_tensor_count(a->info) * sizeof(float);
		if (stream_context)
		{
			int device = ccv_nnc_stream_context_get_device(stream_context);
			hipSetDevice(device);
			hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
			if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_GPU_MEMORY)
				hipMemcpyAsync(b->data.u8, a->data.u8, size, hipMemcpyHostToDevice, stream);
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_CPU_MEMORY)
				hipMemcpyAsync(b->data.u8, a->data.u8, size, hipMemcpyDeviceToHost, stream);
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_CPU_MEMORY)
				hipMemcpyAsync(b->data.u8, a->data.u8, size, hipMemcpyHostToHost, stream);
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_GPU_MEMORY) {
				int device_a = CCV_TENSOR_GET_DEVICE_ID(a->info.type);
				int device_b = CCV_TENSOR_GET_DEVICE_ID(b->info.type);
				if (device_a == device_b)
					hipMemcpyAsync(b->data.u8, a->data.u8, size, hipMemcpyDeviceToDevice, stream);
				else
					hipMemcpyPeerAsync(b->data.u8, device_b, a->data.u8, device_a, size, stream);
			}
		} else {
			if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_GPU_MEMORY)
				hipMemcpy(b->data.u8, a->data.u8, size, hipMemcpyHostToDevice);
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_CPU_MEMORY)
				hipMemcpy(b->data.u8, a->data.u8, size, hipMemcpyDeviceToHost);
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_CPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_CPU_MEMORY)
				hipMemcpy(b->data.u8, a->data.u8, size, hipMemcpyHostToHost);
			else if (CCV_TENSOR_GET_MEMORY(a->info.type) == CCV_TENSOR_GPU_MEMORY && CCV_TENSOR_GET_MEMORY(b->info.type) == CCV_TENSOR_GPU_MEMORY) {
				int device_a = CCV_TENSOR_GET_DEVICE_ID(a->info.type);
				int device_b = CCV_TENSOR_GET_DEVICE_ID(b->info.type);
				if (device_a == device_b)
					hipMemcpy(b->data.u8, a->data.u8, size, hipMemcpyDeviceToDevice);
				else
					hipMemcpyPeer(b->data.u8, device_b, a->data.u8, device_a, size);
			}
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_DATA_TRANSFER_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F;
	registry->tensor_memory = CCV_TENSOR_CPU_MEMORY | CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_data_transfer;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_DATA_TRANSFER_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F;
	registry->tensor_memory = CCV_TENSOR_CPU_MEMORY | CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_data_transfer;
}
