extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

enum {
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_GEMM, // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, // HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_GEMM, // HIPDNN_CONVOLUTION_FWD_ALGO_GEMM
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_DIRECT, // HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT, // HIPDNN_CONVOLUTION_FWD_ALGO_FFT
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT_TILING, // HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_WINOGRAD, // HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_WINOGRAD_NONFUSED, // HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED
	CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_COUNT
};

static int _ccv_nnc_conv_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, const ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 3);
	assert(output_size == 1);
	assert(stream_context);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	int device = ccv_nnc_stream_context_get_device(stream_context);
	hipSetDevice(device);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_filter_descriptor_t w = ccv_nnc_cudnn_get_filter_descriptor(stream_context, (const ccv_nnc_tensor_t*)inputs[1]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t bias = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[2]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	const ccv_nnc_cudnn_convolution_descriptor_t conv = ccv_nnc_cudnn_get_convolution_descriptor(stream_context, hint);

	hipdnnConvolutionFwdAlgo_t algo;
	// Choose an algorithm
	switch (cmd.algorithm)
	{
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_GEMM:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_PRECOMP_GEMM:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_GEMM:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_DIRECT:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_FFT;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT_TILING:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_WINOGRAD:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_WINOGRAD_NONFUSED:
			algo = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
			break;
		default: // -1: Using preferences to find a suitable algorithm
			assert_cudnn(hipdnnGetConvolutionForwardAlgorithm(cudnn, a.descriptor, w.descriptor, conv.descriptor, b.descriptor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));
	}

	size_t workspace_size = 0;
	assert_cudnn(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, a.descriptor, w.descriptor, conv.descriptor, b.descriptor, algo, &workspace_size));
	void* workspace = 0;
	// TODO: If error, return OOM
	if (workspace_size)
		hipMalloc(&workspace, workspace_size);
	const float one = 1, zero = 0;
	assert_cudnn(hipdnnConvolutionForward(cudnn, &one, a.descriptor, a.data.u8, w.descriptor, w.data.u8, conv.descriptor, algo, workspace, workspace_size, &zero, b.descriptor, b.data.u8));
	assert_cudnn(hipdnnAddTensor(cudnn, &one, bias.descriptor, bias.data.u8, &one, b.descriptor, b.data.u8));
	if (workspace)
		hipFreeAsync(workspace, stream);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_filter_descriptor(w);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(bias);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	ccv_nnc_cudnn_deinit_convolution_descriptor(conv);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_conv_forw_autotune(const ccv_nnc_cmd_t cmd, const size_t max_workspace_size, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, const ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 3);
	assert(output_size == 1);
	assert(stream_context);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	int device = ccv_nnc_stream_context_get_device(stream_context);
	hipSetDevice(device);
	void* workmem = 0;
	hipMalloc(&workmem, max_workspace_size);
	if (!workmem)
		return -1;
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_filter_descriptor_t w = ccv_nnc_cudnn_get_filter_descriptor(stream_context, (const ccv_nnc_tensor_t*)inputs[1]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	const ccv_nnc_cudnn_convolution_descriptor_t conv = ccv_nnc_cudnn_get_convolution_descriptor(stream_context, hint);
	int count = 0;
	hipdnnConvolutionFwdAlgoPerf_t perfs[CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_COUNT];
	assert_cudnn(hipdnnFindConvolutionForwardAlgorithmEx(cudnn, a.descriptor, a.data.u8, w.descriptor, w.data.u8, conv.descriptor, b.descriptor, b.data.u8, CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_COUNT, &count, perfs, workmem, max_workspace_size));
	hipFree(workmem);
	int i;
	hipdnnConvolutionFwdAlgo_t algorithm;
	for(i = 0; i < count; i++)
		if ((size_t)perfs[i].memory <= max_workspace_size && perfs[i].status == HIPDNN_STATUS_SUCCESS)
		{
			algorithm = perfs[i].algo;
			break;
		}
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_filter_descriptor(w);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	ccv_nnc_cudnn_deinit_convolution_descriptor(conv);
	switch (algorithm)
	{
		case HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM:
			return CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_GEMM;
		case HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM:
			return CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
		case HIPDNN_CONVOLUTION_FWD_ALGO_GEMM:
			return CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_GEMM;
		case HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT:
			return CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_DIRECT;
		case HIPDNN_CONVOLUTION_FWD_ALGO_FFT:
			return CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT;
		case HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING:
			return CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_FFT_TILING;
		case HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD:
			return CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_WINOGRAD;
		case HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED:
			return CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_WINOGRAD_NONFUSED;
	}
	return -1; // Return the most efficient algorithm, return -1 if cannot find one.
}

enum {
	CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_0, // HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0
	CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_1, // HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1
	CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_FFT, // HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT
	CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_3, // HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_3
	CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_COUNT
};

enum {
	CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_0, // HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0
	CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_1, // HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1
	CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_FFT, // HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT
	CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_FFT_TILING, // HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT_TILING
	CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_FFT_WINOGRAD, // HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD
	CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_COUNT
};

static int _ccv_nnc_conv_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, const ccv_nnc_stream_context_t* const stream_context)
{
	// inputs: gradient, forw prop input, [w]
	// outputs: [output gradient], weight updates, bias updates
	assert((input_size == 2 && output_size == 3) || (input_size == 3 && output_size == 3));
	assert(stream_context);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	int device = ccv_nnc_stream_context_get_device(stream_context);
	hipSetDevice(device);
	const ccv_nnc_cudnn_tensor_view_descriptor_t g = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[1]);
	const ccv_nnc_cudnn_filter_descriptor_t dw = ccv_nnc_cudnn_get_filter_descriptor(stream_context, (const ccv_nnc_tensor_t*)outputs[1]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t bias = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[2]);
	const ccv_nnc_cudnn_convolution_descriptor_t conv = ccv_nnc_cudnn_get_convolution_descriptor(stream_context, hint);

	hipdnnConvolutionBwdFilterAlgo_t filter_algo;
	// Choose an algorithm
	switch (cmd.algorithm % CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_COUNT)
	{
		case CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_0:
			filter_algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_1:
			filter_algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_FFT:
			filter_algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT;
			break;
		case CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_3:
			filter_algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_3;
			break;
		default: // -1: Using preferences to find a suitable algorithm
			assert_cudnn(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn, a.descriptor, g.descriptor, conv.descriptor, dw.descriptor, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &filter_algo));
	}

	size_t workspace_size = 0;
	assert_cudnn(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn, a.descriptor, g.descriptor, conv.descriptor, dw.descriptor, filter_algo, &workspace_size));
	void* workspace = 0;
	// TODO: If error, return OOM
	if (workspace_size)
		hipMalloc(&workspace, workspace_size);
	const float one = 1, zero = 0;
	if ((flags & CCV_NNC_ACCUMULATE_OUTPUT)) // accumulating results to bias and dw
	{
		assert_cudnn(hipdnnConvolutionBackwardBias(cudnn, &one, g.descriptor, g.data.u8, &one, bias.descriptor, bias.data.u8));
		assert_cudnn(hipdnnConvolutionBackwardFilter(cudnn, &one, a.descriptor, a.data.u8, g.descriptor, g.data.u8, conv.descriptor, filter_algo, workspace, workspace_size, &one, dw.descriptor, dw.data.u8));
	} else {
		assert_cudnn(hipdnnConvolutionBackwardBias(cudnn, &one, g.descriptor, g.data.u8, &zero, bias.descriptor, bias.data.u8));
		assert_cudnn(hipdnnConvolutionBackwardFilter(cudnn, &one, a.descriptor, a.data.u8, g.descriptor, g.data.u8, conv.descriptor, filter_algo, workspace, workspace_size, &zero, dw.descriptor, dw.data.u8));
	}
	if (workspace)
		hipFreeAsync(workspace, stream);
	// If h is available, therefore, we need to propagate the gradients back
	if (outputs[0])
	{
		assert(input_size == 3);
		const ccv_nnc_cudnn_filter_descriptor_t w = ccv_nnc_cudnn_get_filter_descriptor(stream_context, (const ccv_nnc_tensor_t*)inputs[2]);
		const ccv_nnc_cudnn_tensor_view_descriptor_t h = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
		hipdnnConvolutionBwdDataAlgo_t data_algo;
		switch (cmd.algorithm / CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_COUNT)
		{
			case CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_0:
				data_algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0;
				break;
			case CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_1:
				data_algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1;
				break;
			case CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_FFT:
				data_algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT;
				break;
			case CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_FFT_TILING:
				data_algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT_TILING;
				break;
			case CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_FFT_WINOGRAD:
				data_algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD;
				break;
			default: // -1: Using preferences to find a suitable algorithm
				assert_cudnn(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn, w.descriptor, g.descriptor, conv.descriptor, h.descriptor, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &data_algo));
		}
		size_t workspace_size = 0;
		assert_cudnn(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn, w.descriptor, g.descriptor, conv.descriptor, h.descriptor, data_algo, &workspace_size));
		void* workspace = 0;
		// TODO: If error, return OOM
		if (workspace_size)
			hipMalloc(&workspace, workspace_size);
		assert_cudnn(hipdnnConvolutionBackwardData(cudnn, &one, w.descriptor, w.data.u8, g.descriptor, g.data.u8, conv.descriptor, data_algo, workspace, workspace_size, &zero, h.descriptor, h.data.u8));
		if (workspace)
			hipFreeAsync(workspace, stream);
		ccv_nnc_cudnn_deinit_filter_descriptor(w);
		ccv_nnc_cudnn_deinit_tensor_view_descriptor(h);
	}
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(g);
	ccv_nnc_cudnn_deinit_filter_descriptor(dw);
	ccv_nnc_cudnn_deinit_convolution_descriptor(conv);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(bias);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_conv_back_autotune(const ccv_nnc_cmd_t cmd, const size_t max_workspace_size, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, const ccv_nnc_stream_context_t* const stream_context)
{
	// inputs: gradient, forw prop input, w
	// outputs:  output gradient, weight updates, bias updates [unused]
	assert(input_size == 3 && output_size == 3);
	assert(stream_context);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	int device = ccv_nnc_stream_context_get_device(stream_context);
	hipSetDevice(device);
	void* workmem = 0;
	hipMalloc(&workmem, max_workspace_size);
	if (!workmem)
		return -1;
	const ccv_nnc_cudnn_tensor_view_descriptor_t g = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)inputs[1]);
	const ccv_nnc_cudnn_filter_descriptor_t w = ccv_nnc_cudnn_get_filter_descriptor(stream_context, (const ccv_nnc_tensor_t*)inputs[2]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t h = ccv_nnc_cudnn_get_tensor_view_descriptor(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	const ccv_nnc_cudnn_filter_descriptor_t dw = ccv_nnc_cudnn_get_filter_descriptor(stream_context, (const ccv_nnc_tensor_t*)outputs[1]);
	const ccv_nnc_cudnn_convolution_descriptor_t conv = ccv_nnc_cudnn_get_convolution_descriptor(stream_context, hint);
	int count = 0;
	hipdnnConvolutionBwdFilterAlgoPerf_t filter_perfs[CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_COUNT];
	assert_cudnn(hipdnnFindConvolutionBackwardFilterAlgorithmEx(cudnn, a.descriptor, a.data.u8, g.descriptor, g.data.u8, conv.descriptor, dw.descriptor, dw.data.u8, CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_COUNT, &count, filter_perfs, workmem, max_workspace_size));
	int i;
	hipdnnConvolutionBwdFilterAlgo_t filter_algorithm;
	for(i = 0; i < count; i++)
		if ((size_t)filter_perfs[i].memory <= max_workspace_size && filter_perfs[i].status == HIPDNN_STATUS_SUCCESS)
		{
			filter_algorithm = filter_perfs[i].algo;
			break;
		}
	hipdnnConvolutionBwdDataAlgoPerf_t data_perfs[CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_COUNT];
	assert_cudnn(hipdnnFindConvolutionBackwardDataAlgorithmEx(cudnn, w.descriptor, w.data.u8, g.descriptor, g.data.u8, conv.descriptor, h.descriptor, h.data.u8, CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_COUNT, &count, data_perfs, workmem, max_workspace_size));
	hipdnnConvolutionBwdDataAlgo_t data_algorithm;
	for(i = 0; i < count; i++)
		if ((size_t)data_perfs[i].memory <= max_workspace_size && data_perfs[i].status == HIPDNN_STATUS_SUCCESS)
		{
			data_algorithm = data_perfs[i].algo;
			break;
		}
	hipFree(workmem);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(g);
	ccv_nnc_cudnn_deinit_filter_descriptor(w);
	ccv_nnc_cudnn_deinit_filter_descriptor(dw);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(h);
	ccv_nnc_cudnn_deinit_convolution_descriptor(conv);
	int filter = -1, data = -1;
	switch (filter_algorithm)
	{
		case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0:
			filter = CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_0;
			break;
		case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1:
			filter = CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_1;
			break;
		case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_FFT:
			filter = CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_FFT;
			break;
		case HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_3:
			filter = CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_3;
			break;
	}
	switch (data_algorithm)
	{
		case HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0:
			data = CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_0;
			break;
		case HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1:
			data = CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_1;
			break;
		case HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT:
			data = CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_FFT;
			break;
		case HIPDNN_CONVOLUTION_BWD_DATA_ALGO_FFT_TILING:
			data = CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_FFT_TILING;
			break;
		case HIPDNN_CONVOLUTION_BWD_DATA_ALGO_WINOGRAD:
			data = CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_FFT_WINOGRAD;
			break;
	}
	return data * CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_COUNT + filter;
}
#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_CONVOLUTION_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = CCV_NNC_CMD_CUDNN_CONV_FWD_ALGO_COUNT;
	registry->exec = _ccv_nnc_conv_forw;
	registry->autotune = _ccv_nnc_conv_forw_autotune;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_CONVOLUTION_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_32F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = CCV_NNC_CMD_CUDNN_CONV_BWD_DATA_ALGO_COUNT * CCV_NNC_CMD_CUDNN_CONV_BWD_FILTER_ALGO_COUNT;
	registry->exec = _ccv_nnc_conv_back;
	registry->autotune = _ccv_nnc_conv_back_autotune;
#endif
}
